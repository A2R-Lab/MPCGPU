#include "hip/hip_runtime.h"
#include <fstream>
#include <vector>
#include <sstream>
#include <iostream>
#include <tuple>
#include <filesystem>
#include "dynamics/rbd_plant.cuh"
#include "settings.cuh"
#include "utils/experiment.cuh"
#include "gpu_pcg.cuh"
#include "../include/pcg/sqp_n.cuh"


int main(int argc, char **argv){
    constexpr uint32_t state_size = grid::NUM_JOINTS*2;
    constexpr uint32_t control_size = grid::NUM_JOINTS;
    constexpr uint32_t knot_points = KNOT_POINTS;
    const float timestep = .015625;
	const uint32_t traj_len = (state_size+control_size)*knot_points-control_size;
	const uint32_t solve_count = atoi(argv[1]);
	

    checkPcgOccupancy<linsys_t>((void *) pcg<linsys_t, state_size, knot_points>, PCG_NUM_THREADS, state_size, knot_points);    
	void *d_dynmem = gato_plant::initializeDynamicsConstMem<linsys_t>();

    linsys_t *d_eePos_traj, *d_xu_traj;
	linsys_t rho = 1e-3;

	// vars for recording data
    std::vector<std::vector<int>> linsys_iters;
    std::vector<std::vector<double>> linsys_times;
    double sqp_time;
    uint32_t sqp_iters;
    std::vector<bool> sqp_exits;
    std::vector<std::vector<bool>> linsys_exits;

	std::tuple<std::vector<std::vector<int>>, // pcg_iter_vec
				std::vector<std::vector<double>>, // linsys_time_vec
				double, // sqp_solve_time for all of the problems
				uint32_t, // sqp_iter, max iteration for all of the problems
				std::vector<bool>, // sqp_time_exit
				std::vector<std::vector<bool>>> // pcg_exit_vec
				sqp_stats; 


	uint32_t num_exit_vals = 5;
	linsys_t pcg_exit_vals[num_exit_vals];
	pcg_exit_vals[0] = 5e-6;
	pcg_exit_vals[1] = 7.5e-6;
	pcg_exit_vals[2] = 5e-6;
	pcg_exit_vals[3] = 2.5e-6;
	pcg_exit_vals[4] = 1e-6;
	

	linsys_t pcg_exit_tol = pcg_exit_vals[0];
	std::vector<toplevel_return_type> current_results;

	pcg_config<linsys_t> config;
	config.pcg_block = PCG_NUM_THREADS;
	config.pcg_exit_tol = pcg_exit_tol;
	config.pcg_max_iter = PCG_MAX_ITER;

	// read in traj
	auto eePos_traj2d = readCSVToVecVec<linsys_t>("examples/trajfiles/0_0_eepos.traj");
	auto xu_traj2d = readCSVToVecVec<linsys_t>("examples/trajfiles/0_0_traj.csv");

	std::vector<linsys_t> h_eePos_traj;
	for (const auto& vec : eePos_traj2d) {
		h_eePos_traj.insert(h_eePos_traj.end(), vec.begin(), vec.end());
	}
	std::vector<linsys_t> h_xu_traj;
	for (const auto& xu_vec : xu_traj2d) {
		h_xu_traj.insert(h_xu_traj.end(), xu_vec.begin(), xu_vec.end());
	}

	uint32_t lambda_size = state_size * knot_points * solve_count * sizeof(linsys_t);
	uint32_t xu_size = traj_len * sizeof(linsys_t);
	uint32_t eePos_size = 6 * knot_points * sizeof(linsys_t);
	linsys_t *d_lambda;

	gpuErrchk(hipMalloc(&d_eePos_traj, solve_count * eePos_size));
	gpuErrchk(hipMalloc(&d_xu_traj, solve_count * xu_size));
	gpuErrchk(hipMalloc(&d_lambda, lambda_size));

	for (uint32_t i = 0; i < 100; i++) {
		sqpSolvePcg<linsys_t>(1, state_size, control_size, knot_points, timestep, d_eePos_traj, d_lambda, d_xu_traj, d_dynmem, config, rho, 1e-3);
	}
	
	gpuErrchk(hipMemset(d_lambda, 0, lambda_size));
	gpuErrchk(hipMemcpy(d_xu_traj, h_xu_traj.data() + xu_size, xu_size, hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_eePos_traj,  h_eePos_traj.data() + eePos_size, eePos_size, hipMemcpyHostToDevice));

	//  copy to to the other problems
	for (uint32_t i = 1; i < solve_count; i++) {
		gpuErrchk(hipMemcpy(d_xu_traj + i * traj_len, d_xu_traj, xu_size, hipMemcpyDeviceToDevice));
		gpuErrchk(hipMemcpy(d_eePos_traj + i * 6 * knot_points, d_eePos_traj, eePos_size, hipMemcpyDeviceToDevice));
	}
	gpuErrchk(hipDeviceSynchronize());

	sqp_stats = sqpSolvePcg<linsys_t>(solve_count, state_size, control_size, knot_points, timestep, d_eePos_traj, d_lambda, d_xu_traj, d_dynmem, config, rho, 1e-3);
	gpuErrchk(hipDeviceSynchronize());
	linsys_t h_xu[traj_len * solve_count];
	linsys_t h_lambda[lambda_size / sizeof(linsys_t)];
	linsys_t h_eePos[6 * knot_points * solve_count];
	gpuErrchk(hipMemcpy(h_xu, d_xu_traj, xu_size * solve_count, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_eePos, d_eePos_traj, eePos_size * solve_count, hipMemcpyDeviceToHost));
	gpuErrchk(hipMemcpy(h_lambda, d_lambda, lambda_size, hipMemcpyDeviceToHost));


	linsys_iters = std::get<0>(sqp_stats);
	linsys_times = std::get<1>(sqp_stats);
	sqp_time = std::get<2>(sqp_stats);
	sqp_iters = std::get<3>(sqp_stats);
	sqp_exits = std::get<4>(sqp_stats);
	linsys_exits = std::get<5>(sqp_stats);

	printf("sqp time: %f\n", sqp_time);

	for (uint32_t i = 0; i < solve_count; i++) {
		printf("problem: %d\n", i);
		for (uint32_t j = 0; j < state_size * knot_points; j++) {
			printf("%f, ", h_lambda[j + i * state_size * knot_points]);
		}
		printf("\n");
	}

	for (uint32_t i = 0; i < solve_count; i++) {
		printf("problem: %d\n", i);
		for (uint32_t j = 0; j < 6 * knot_points; j++) {
			printf("%f, ", h_eePos[j + i * 6 * knot_points]);
		}
		printf("\n");
	}

	gpuErrchk(hipFree(d_xu_traj));
	gpuErrchk(hipFree(d_eePos_traj));
	gpuErrchk(hipFree(d_lambda));
	gpuErrchk(hipPeekAtLastError());
    return 0;
}
